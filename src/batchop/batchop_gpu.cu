#include "hip/hip_runtime.h"

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <hipsolver.h>

static char const * const errInputId = "batchop_gpu:InvalidInput";
static char const * const errCudaId = "batchop_gpu:CudaError";
static char const * const errCudaMsg = "batchop_gpu encountered a cuda error.";

static bool solverInitialized = false;
static hipblasHandle_t cublasHandle = NULL;
static hipsolverHandle_t cusolverHandle = NULL;
static hipsolverSyevjInfo_t syevj_params = NULL;
static hipsolverGesvdjInfo_t gesvdj_params = NULL;

static mxArray * gpuCanary;

static void uninit();

static bool init() {
    if (!solverInitialized || !mxGPUIsValidGPUData(gpuCanary)) {
        // Initialize the MATLAB GPU API if not already initialized.
        if (mxInitGPU() != MX_GPU_SUCCESS) {
            return false;
        }

        const size_t one = 1;
        mxGPUArray * canary = mxGPUCreateGPUArray(1, &one, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        gpuCanary = mxGPUCreateMxArrayOnGPU(canary);
        mxGPUDestroyGPUArray(canary);
        mexMakeArrayPersistent(gpuCanary);

        hipblasStatus_t blastat = hipblasCreate(&cublasHandle);
        if (blastat != HIPBLAS_STATUS_SUCCESS) {
            return false;
        }
        
        hipsolverStatus_t status = hipsolverDnCreate(&cusolverHandle);
        if (status != HIPSOLVER_STATUS_SUCCESS) {
            hipblasDestroy(cublasHandle);
            return false;
        }

        status = hipsolverDnCreateSyevjInfo(&syevj_params);
        if (status != HIPSOLVER_STATUS_SUCCESS) {
            hipsolverDnDestroy(cusolverHandle);
            hipblasDestroy(cublasHandle);
            return false;
        }

        status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
        if (status != HIPSOLVER_STATUS_SUCCESS) {
            hipsolverDnDestroySyevjInfo(syevj_params);
            hipsolverDnDestroy(cusolverHandle);
            hipblasDestroy(cublasHandle);
            return false;
        }
        
        solverInitialized = true;
        mexAtExit(uninit);
    }
    return true;
}

static void uninit() {
    if (solverInitialized) {
        if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);
        if (syevj_params) hipsolverDnDestroySyevjInfo(syevj_params);
        if (cusolverHandle) hipsolverDnDestroy(cusolverHandle);
        if (cublasHandle) hipblasDestroy(cublasHandle);
        mxDestroyArray(gpuCanary);
    }
}

static
mxGPUArray *
doMult(const mxGPUArray *A, const size_t * dimA, const std::string& transpA,
       const mxGPUArray *B, const size_t * dimB,  const std::string& transpB,
       size_t nPages) {
    size_t dimA0 = dimA[0], dimB0 = dimB[0];
    size_t m = dimA[0], r1 = dimA[1], r2 = dimB[0], n = dimB[1];
    hipblasOperation_t opA = HIPBLAS_OP_N, opB = HIPBLAS_OP_N;
    if (transpA == "T") {
        m = dimA[1];
        r1 = dimA[0];
        opA = HIPBLAS_OP_T;
    }
    if (transpB == "T") {
        r2 = dimB[1];
        n = dimB[0];
        opB = HIPBLAS_OP_T;
    }

    if (r1 != r2) {
        mexErrMsgIdAndTxt(errInputId, "Inner dimensions do not match.");
    }

    size_t dimC[] = {m, n, nPages};
    mxGPUArray *C = mxGPUCreateGPUArray(3, dimC, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Scalar required by cublas.
    const double one = 1.0, zero = 0.0;

    // Compute A * B
    hipblasStatus_t status = hipblasDgemmStridedBatched(
        cublasHandle, opA, opB, m, n, r1,
        &one, static_cast<const double *>(mxGPUGetDataReadOnly(A)), dimA0, m * r1,
        static_cast<const double *>(mxGPUGetDataReadOnly(B)), dimB0, r1 * n,
        &zero, static_cast<double *>(mxGPUGetData(C)), m, m * n,
        nPages);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    return C;
}

static
thrust::tuple<mxGPUArray *, mxGPUArray *>
doCholesky(const mxGPUArray *A, const size_t * dimA, size_t nPages) {
    size_t m = dimA[0];
    if (dimA[1] != m) {
        mexErrMsgIdAndTxt(errInputId, "Matrices are not square.");
    }

    // Copy A into L. This will be overwritten with the Cholesky factor in place.
    mxGPUArray *L = mxGPUCopyGPUArray(A);

    // Create array to store factorization status.
    mxGPUArray *info = mxGPUCreateGPUArray(1, &nPages, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Create array of pointers into pages of L.
    thrust::device_vector<size_t> ptrL(nPages, m * m * sizeof(double));
    thrust::exclusive_scan(ptrL.begin(), ptrL.end(), ptrL.begin(),
                           reinterpret_cast<size_t>(mxGPUGetData(L)));

    // Factor L = A in place.
    hipsolverStatus_t status = hipsolverDnDpotrfBatched(
        cusolverHandle, HIPBLAS_FILL_MODE_LOWER, m,
        reinterpret_cast<double **>(ptrL.data().get()), m,
        static_cast<int *>(mxGPUGetData(info)),
        nPages);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    return thrust::make_tuple(L, info);
}

static
mxGPUArray *
doCholsolve(const mxGPUArray * L, const size_t * dimL,
            const mxGPUArray * B, const size_t * dimB,
            size_t nPages) {
    size_t m = dimL[0], n = dimB[1];
    if (dimL[0] != m || dimB[0] != m) {
        mexErrMsgIdAndTxt(errInputId, "Dimensions do not match.");
    }
    
    // Copy B into X. This will be overwritten with the solution.
    mxGPUArray *X = mxGPUCopyGPUArray(B);

    // Create array of pointers into pages of L.
    thrust::device_vector<size_t> ptrL(nPages, m * m * sizeof(double));
    thrust::exclusive_scan(ptrL.begin(), ptrL.end(), ptrL.begin(),
                           reinterpret_cast<size_t>(mxGPUGetDataReadOnly(L)));


    // Create array of pointers into pages of X.
    thrust::device_vector<size_t> ptrX(nPages, m * n * sizeof(double));
    thrust::exclusive_scan(ptrX.begin(), ptrX.end(), ptrX.begin(),
                           reinterpret_cast<size_t>(mxGPUGetData(X)));


    if (n == 1) {
        // Dummy array to store solve status.
        size_t one = 1;
        mxGPUArray *info = mxGPUCreateGPUArray(1, &one, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

        hipsolverStatus_t status = hipsolverDnDpotrsBatched(
            cusolverHandle, HIPBLAS_FILL_MODE_LOWER, m, n,
            reinterpret_cast<double **>(ptrL.data().get()), m,
            reinterpret_cast<double **>(ptrX.data().get()), m,
            static_cast<int *>(mxGPUGetData(info)),
            nPages);
        if (status != HIPSOLVER_STATUS_SUCCESS) {
            mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
        }

        mxGPUDestroyGPUArray(info);
    } else {
        double one = 1.0;
        
        // Compute X = L \ X
        hipblasStatus_t status = hipblasDtrsmBatched(
            cublasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
            m, n,
            &one, reinterpret_cast<double **>(ptrL.data().get()), m,
            reinterpret_cast<double **>(ptrX.data().get()), m,
            nPages);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
        }

        // Compute X = L' \ X = L' \ (L \ B)
        status = hipblasDtrsmBatched(
            cublasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
            m, n,
            &one, reinterpret_cast<double **>(ptrL.data().get()), m,
            reinterpret_cast<double **>(ptrX.data().get()), m,
            nPages);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
        }
    }

    return X;
}

static
mxGPUArray *
doCholCong(const mxGPUArray * L, const size_t * dimL,
           const mxGPUArray * B, const size_t * dimB,
           size_t nPages) {
    size_t m = dimL[0];
    if (dimL[1] != m || dimB[0] != m || dimB[1] != m) {
        mexErrMsgIdAndTxt(errInputId, "Dimensions do not match.");
    }

    // Copy B into X. This will be overwritten with the solution.
    mxGPUArray *X = mxGPUCopyGPUArray(B);

    // Create array of pointers into pages of L.
    thrust::device_vector<size_t> ptrL(nPages, m * m * sizeof(double));
    thrust::exclusive_scan(ptrL.begin(), ptrL.end(), ptrL.begin(),
                           reinterpret_cast<size_t>(mxGPUGetDataReadOnly(L)));


    // Create array of pointers into pages of X.
    thrust::device_vector<size_t> ptrX(nPages, m * m * sizeof(double));
    thrust::exclusive_scan(ptrX.begin(), ptrX.end(), ptrX.begin(),
                           reinterpret_cast<size_t>(mxGPUGetData(X)));

    // Scalar required by cublas.
    double one = 1.0;

    // Compute X = L \ X
    hipblasStatus_t status = hipblasDtrsmBatched(
        cublasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
        m, m,
        &one, reinterpret_cast<double **>(ptrL.data().get()), m,
        reinterpret_cast<double **>(ptrX.data().get()), m,
        nPages);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    // Compute X = X / L' = L \ B / L'
    status = hipblasDtrsmBatched(
        cublasHandle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
        m, m,
        &one, reinterpret_cast<double **>(ptrL.data().get()), m,
        reinterpret_cast<double **>(ptrX.data().get()), m,
        nPages);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    return X;
}

static
thrust::tuple<mxGPUArray *, mxGPUArray *>
doLS(const mxGPUArray * A, const size_t * dimA,
     const mxGPUArray * B, const size_t * dimB,
     size_t nPages) {
    size_t m = dimA[0], r = dimA[1], n = dimB[1];
    if (dimB[0] != m) {
        mexErrMsgIdAndTxt(errInputId, "Dimensions do not match.");
    }

    size_t dimX[] = {r, n, nPages};
    mxGPUArray * X = mxGPUCreateGPUArray(3, dimX, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Copy A and B. Both will be overwritten
    mxGPUArray * Ac = mxGPUCopyGPUArray(A);
    mxGPUArray * Bc = mxGPUCopyGPUArray(B);

    // Create array of pointers into pages of Ac and Bc.
    thrust::device_vector<size_t> ptrAc(nPages, m * r * sizeof(double));
    thrust::exclusive_scan(ptrAc.begin(), ptrAc.end(), ptrAc.begin(),
                           reinterpret_cast<size_t>(mxGPUGetData(Ac)));
    thrust::device_vector<size_t> ptrBc(nPages, m * n * sizeof(double));
    thrust::exclusive_scan(ptrBc.begin(), ptrBc.end(), ptrBc.begin(),
                           reinterpret_cast<size_t>(mxGPUGetData(Bc)));

    mxGPUArray * info = mxGPUCreateGPUArray(1, &nPages, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    int valid;
    hipblasStatus_t status = hipblasDgelsBatched(
        cublasHandle, HIPBLAS_OP_N, m, r, n,
        reinterpret_cast<double **>(ptrAc.data().get()), m,
        reinterpret_cast<double **>(ptrBc.data().get()), m,
        &valid, static_cast<int *>(mxGPUGetData(info)), nPages
    );
    if (status != HIPBLAS_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    // Copy solution to X
    const double zero = 0.0, one = 1.0;
    status = hipblasDgeam(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, r, n * nPages,
        &one, static_cast<const double *>(mxGPUGetDataReadOnly(Bc)), m,
        &zero, static_cast<const double *>(mxGPUGetDataReadOnly(X)), r,
        static_cast<double *>(mxGPUGetData(X)), r
    );
    if (status != HIPBLAS_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    mxGPUDestroyGPUArray(Ac);
    mxGPUDestroyGPUArray(Bc);
    return thrust::make_tuple(X, info);
}

static
thrust::tuple<mxGPUArray *, mxGPUArray *, mxGPUArray *>
doEig(const mxGPUArray * A, const size_t * dimA,
      hipsolverEigMode_t eigmode, size_t nPages) {
    size_t m = dimA[0];
    if (dimA[1] != m) {
        mexErrMsgIdAndTxt(errInputId, "Matrices are not square.");
    }

    // Copy A into Q. This will be overwritten with the eigenvectors.
    mxGPUArray *Q = mxGPUCopyGPUArray(A);

    // Create array to store eigenvalues.
    size_t dimE[] = {m, nPages};
    mxGPUArray *E = mxGPUCreateGPUArray(2, dimE, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Create workspace
    int workSize;
    hipsolverStatus_t status = hipsolverDnDsyevjBatched_bufferSize(
        cusolverHandle, eigmode, HIPBLAS_FILL_MODE_LOWER, m,                
        static_cast<const double *>(mxGPUGetData(Q)), m,              
        static_cast<const double *>(mxGPUGetData(E)),
        &workSize, syevj_params, nPages);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }
    size_t workSize2 = workSize;
    mxGPUArray *work = mxGPUCreateGPUArray(1, &workSize2, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Create array to store eigendecomposition status.
    mxGPUArray *info = mxGPUCreateGPUArray(1, &nPages, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Do factorization
    status = hipsolverDnDsyevjBatched(
        cusolverHandle, eigmode, HIPBLAS_FILL_MODE_LOWER, m,                
        static_cast<double *>(mxGPUGetData(Q)), m,              
        static_cast<double *>(mxGPUGetData(E)),
        static_cast<double *>(mxGPUGetData(work)), workSize,
        static_cast<int *>(mxGPUGetData(info)),
        syevj_params, nPages);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    mxGPUDestroyGPUArray(work);

    return thrust::make_tuple(E, Q, info);
}

static
thrust::tuple<mxGPUArray *, mxGPUArray *, mxGPUArray *, mxGPUArray *>
doSVD(const mxGPUArray *A, const size_t * dimA, size_t rank, size_t nPages) {
    size_t m = dimA[0], n = dimA[1];

    // Copy A. This will be overwritten during the computation.
    mxGPUArray *Ac = mxGPUCopyGPUArray(A);

    // Create arrays to store singular values and vectors
    size_t dimS[] = {std::min(m, n), nPages};
    size_t dimU[] = {m, m, nPages};
    size_t dimV[] = {n, n, nPages};
    mxGPUArray *S = mxGPUCreateGPUArray(2, dimS, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray *U = mxGPUCreateGPUArray(3, dimU, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray *V = mxGPUCreateGPUArray(3, dimV, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Create workspace
    int workSize;
    hipsolverStatus_t status = hipsolverDnDgesvdjBatched_bufferSize(
        cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, m, n,                
        static_cast<const double *>(mxGPUGetData(Ac)), m,              
        static_cast<const double *>(mxGPUGetData(S)), 
        static_cast<const double *>(mxGPUGetData(U)), m,              
        static_cast<const double *>(mxGPUGetData(V)), n,              
        &workSize,
        gesvdj_params, nPages);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }
    size_t workSize2 = workSize;
    mxGPUArray *work = mxGPUCreateGPUArray(1, &workSize2, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Create array to store SVD status.
    mxGPUArray *info = mxGPUCreateGPUArray(1, &nPages, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Do factorization
    status = hipsolverDnDgesvdjBatched(
        cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, m, n,                
        static_cast<double *>(mxGPUGetData(Ac)), m,              
        static_cast<double *>(mxGPUGetData(S)),
        static_cast<double *>(mxGPUGetData(U)), m,              
        static_cast<double *>(mxGPUGetData(V)), n,
        static_cast<double *>(mxGPUGetData(work)), workSize,
        static_cast<int *>(mxGPUGetData(info)),
        gesvdj_params, nPages);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    mxGPUDestroyGPUArray(work);
    mxGPUDestroyGPUArray(Ac);

    if (rank < m && rank < n) {
        // Copy partial SVD
        size_t dimSr[] = {rank, nPages};
        size_t dimUr[] = {m, rank, nPages};
        size_t dimVr[] = {n, rank, nPages};
        mxGPUArray *Sr = mxGPUCreateGPUArray(2, dimSr, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        mxGPUArray *Ur = mxGPUCreateGPUArray(3, dimUr, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        mxGPUArray *Vr = mxGPUCreateGPUArray(3, dimVr, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

        hipError_t custat = hipMemcpy2D(
            mxGPUGetData(Sr), rank * sizeof(double),
            mxGPUGetData(S), std::min(m, n) * sizeof(double),
            rank * sizeof(double), nPages, hipMemcpyDeviceToDevice);
        if (custat != hipSuccess) {
            mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
        }

        custat = hipMemcpy2D(
            mxGPUGetData(Ur), m * rank * sizeof(double),
            mxGPUGetData(U), m * m * sizeof(double),
            m * rank * sizeof(double), nPages, hipMemcpyDeviceToDevice);
        if (custat != hipSuccess) {
            mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
        }

        custat = hipMemcpy2D(
            mxGPUGetData(Vr), n * rank * sizeof(double),
            mxGPUGetData(V), n * n * sizeof(double),
            n * rank * sizeof(double), nPages, hipMemcpyDeviceToDevice);
        if (custat != hipSuccess) {
            mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
        }

        mxGPUDestroyGPUArray(U);
        mxGPUDestroyGPUArray(S);
        mxGPUDestroyGPUArray(V);
        return thrust::make_tuple(Ur, Sr, Vr, info);
    }

    return thrust::make_tuple(U, S, V, info);
}

#if false
static
thrust::tuple<mxGPUArray *, mxGPUArray *, mxGPUArray *, mxGPUArray *>
doSVDApprox(const mxGPUArray *A, int rank) {
    const mwSize nDimA = mxGPUGetNumberOfDimensions(A);
    const mwSize *dimA = mxGPUGetDimensions(A);
    if (nDimA != 3) {
        mexErrMsgIdAndTxt(errInputId, errInputMsg);
    }
    size_t k = dimA[0];
    size_t l = dimA[1];
    size_t n = dimA[2];
    mxFree((void *)dimA);

    // Create arrays to store subset of singular values and vectors
    size_t dimS[] = {rank, n};
    size_t dimU[] = {k, rank, n};
    size_t dimV[] = {l, rank, n};
    mxGPUArray *S = mxGPUCreateGPUArray(2, dimS, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray *U = mxGPUCreateGPUArray(3, dimU, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray *V = mxGPUCreateGPUArray(3, dimV, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Create workspace
    int workSize;
    hipsolverStatus_t status = hipsolverDnDgesvdaStridedBatched_bufferSize(
        cusolverHandle,
        HIPSOLVER_EIG_MODE_VECTOR,
        rank,
        k,                
        l,                
        static_cast<const double *>(mxGPUGetDataReadOnly(A)),
        k,
        k * l,
        static_cast<const double *>(mxGPUGetData(S)),
        rank,
        static_cast<const double *>(mxGPUGetData(U)),
        k,
        k * rank,
        static_cast<const double *>(mxGPUGetData(V)),
        l,
        l * rank,
        &workSize,
        n);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }
    size_t workSize2 = workSize;
    mxGPUArray *work = mxGPUCreateGPUArray(1, &workSize2, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Create array to store SVD status.
    mxGPUArray *info = mxGPUCreateGPUArray(1, &n, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // Do factorization
    status = hipsolverDnDgesvdaStridedBatched(
        cusolverHandle,
        HIPSOLVER_EIG_MODE_VECTOR,
        rank,
        k,                
        l,                
        static_cast<const double *>(mxGPUGetDataReadOnly(A)),      
        k,
        k * l,
        static_cast<double *>(mxGPUGetData(S)),
        rank,
        static_cast<double *>(mxGPUGetData(U)),
        k,
        k * rank,
        static_cast<double *>(mxGPUGetData(V)),
        l,
        l * rank,
        static_cast<double *>(mxGPUGetData(work)),
        workSize,
        static_cast<int *>(mxGPUGetData(info)),
        NULL,
        n);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    // Clean up
    mxGPUDestroyGPUArray(work);

    return thrust::make_tuple(U, S, V, info);
}
#endif

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    // Initialize solver
    if (!init()) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    if (nrhs < 2) {
        mexErrMsgIdAndTxt(errInputId, "Must provide an operation and input matrix.");
    }

    // Get string indicating which algorithm to perform.
    char * algStr_ptr = mxArrayToUTF8String(prhs[0]);
    const std::string algStr = algStr_ptr;
    mxFree(algStr_ptr);

	// Throw an error if the input is not a GPU array.
    if (!mxIsGPUArray(prhs[1]) || !mxGPUIsValidGPUData(prhs[1])) {
        mexErrMsgIdAndTxt(errInputId, "Inputs to batchop_gpu must be of type double gpuArray.");
    }

    // Unwrap input to an mxGPUArray (must be real double).
    const mxGPUArray *A = mxGPUCreateFromMxArray(prhs[1]);
    if (mxGPUGetClassID(A) != mxDOUBLE_CLASS || mxGPUGetComplexity(A) != mxREAL) {
        mexErrMsgIdAndTxt(errInputId, "Inputs to batchop_gpu must be of type double gpuArray.");
    }

    size_t nDimA = mxGPUGetNumberOfDimensions(A);
    if (nDimA < 2 || nDimA > 3) {
        mexErrMsgIdAndTxt(errInputId, "batchop operates on 2D or 3D arrays only.");
    }
    const size_t * dimA = mxGPUGetDimensions(A);
    size_t nPages = (nDimA == 2) ? 1 : dimA[2];

    if (algStr == "chol") {
        mxGPUArray *L, *info;
        thrust::tie(L, info) = doCholesky(A, dimA, nPages);
        plhs[0] = mxGPUCreateMxArrayOnGPU(L);
        plhs[1] = mxGPUCreateMxArrayOnGPU(info);
        mxGPUDestroyGPUArray(L);
        mxGPUDestroyGPUArray(info);
    } else if (algStr == "eigval") {
        mxGPUArray *E, *Q, *info;
        thrust::tie(E, Q, info) = doEig(A, dimA, HIPSOLVER_EIG_MODE_NOVECTOR, nPages);
        plhs[0] = mxGPUCreateMxArrayOnGPU(E);
        plhs[1] = mxGPUCreateMxArrayOnGPU(info);
        mxGPUDestroyGPUArray(E);
        mxGPUDestroyGPUArray(Q);
        mxGPUDestroyGPUArray(info);
    } else if (algStr == "eig") {
        mxGPUArray *E, *Q, *info;
        thrust::tie(E, Q, info) = doEig(A, dimA, HIPSOLVER_EIG_MODE_VECTOR, nPages);
        plhs[0] = mxGPUCreateMxArrayOnGPU(E);
        plhs[1] = mxGPUCreateMxArrayOnGPU(Q);
        plhs[2] = mxGPUCreateMxArrayOnGPU(info);
        mxGPUDestroyGPUArray(E);
        mxGPUDestroyGPUArray(Q);
        mxGPUDestroyGPUArray(info);
    } else if (algStr == "svd") {
        if (nrhs != 3 || !mxIsScalar(prhs[2])) {
            mexErrMsgIdAndTxt(errInputId, "Must provide desired rank.");
        }

        mxGPUArray *U, *S, *V, *info;
        thrust::tie(U, S, V, info) = doSVD(A, dimA, static_cast<size_t>(mxGetScalar(prhs[2])), nPages);
        plhs[0] = mxGPUCreateMxArrayOnGPU(U);
        plhs[1] = mxGPUCreateMxArrayOnGPU(S);
        plhs[2] = mxGPUCreateMxArrayOnGPU(V);
        plhs[3] = mxGPUCreateMxArrayOnGPU(info);
        mxGPUDestroyGPUArray(U);
        mxGPUDestroyGPUArray(S);
        mxGPUDestroyGPUArray(V);
        mxGPUDestroyGPUArray(info);
    } else { // Binary operations
        if (nrhs < 3) {
            mexErrMsgIdAndTxt(errInputId, "Binary operation requires two inputs.");
        } else if (!mxIsGPUArray(prhs[2]) || !mxGPUIsValidGPUData(prhs[2])) {
            mexErrMsgIdAndTxt(errInputId, "Inputs to batchop_gpu must be of type double gpuArray.");
        }
        const mxGPUArray *B = mxGPUCreateFromMxArray(prhs[2]);
        if (mxGPUGetClassID(B) != mxDOUBLE_CLASS || mxGPUGetComplexity(B) != mxREAL) {
            mexErrMsgIdAndTxt(errInputId, "Inputs to batchop_gpu must be of type double gpuArray.");
        }

        size_t nDimB = mxGPUGetNumberOfDimensions(B);
        const size_t * dimB = mxGPUGetDimensions(B);
        if (nDimB != nDimA || (nPages > 1 && dimB[2] != nPages)) {
            mexErrMsgIdAndTxt(errInputId, "Dimensions do not match.");
        }

        if (algStr == "cholcong") {
            // A actually stores Cholesky factor L. Compute L \ B / L'
            mxGPUArray *X = doCholCong(A, dimA, B, dimB, nPages);
            plhs[0] = mxGPUCreateMxArrayOnGPU(X);
            mxGPUDestroyGPUArray(X);
        } else if (algStr == "cholsolve") {
            // A actually stores Cholesky factor L. Compute L' \ (L \ B)
            mxGPUArray *X = doCholsolve(A, dimA, B, dimB, nPages);
            plhs[0] = mxGPUCreateMxArrayOnGPU(X);
            mxGPUDestroyGPUArray(X);
        } else if (algStr == "leastsq") {
            mxGPUArray *X, *info;
            thrust::tie(X, info) = doLS(A, dimA, B, dimB, nPages);
            plhs[0] = mxGPUCreateMxArrayOnGPU(X);
            plhs[1] = mxGPUCreateMxArrayOnGPU(info);
            mxGPUDestroyGPUArray(X);
            mxGPUDestroyGPUArray(info);
        } else if (algStr == "mult") {
            std::string transpA = "N";
            if (nrhs >= 4) {
                if (!mxIsChar(prhs[3])) {
                    mexErrMsgIdAndTxt(errInputId, "Usage: batchop('mult', A, B, ['N' or 'T', 'N' or 'T'])");
                }
                char * transpA_ptr = mxArrayToUTF8String(prhs[3]);
                transpA = transpA_ptr;
                mxFree(transpA_ptr);
                if (transpA != "N" && transpA != "T") {
                    mexErrMsgIdAndTxt(errInputId, "Usage: batchop('mult', A, B, ['N' or 'T', 'N' or 'T'])");
                }
            }

            std::string transpB = "N";
            if (nrhs >= 5) {
                if (!mxIsChar(prhs[4])) {
                    mexErrMsgIdAndTxt(errInputId, "Usage: batchop('mult', A, B, ['N' or 'T', 'N' or 'T'])");
                }
                char * transpB_ptr = mxArrayToUTF8String(prhs[4]);
                transpB = transpB_ptr;
                mxFree(transpB_ptr);
                if (transpB != "N" && transpB != "T") {
                    mexErrMsgIdAndTxt(errInputId, "Usage: batchop('mult', A, B, ['N' or 'T', 'N' or 'T'])");
                }
            }
            mxGPUArray *C = doMult(A, dimA, transpA, B, dimB, transpB, nPages);
            plhs[0] = mxGPUCreateMxArrayOnGPU(C);
            mxGPUDestroyGPUArray(C);
        } else {
            mexErrMsgIdAndTxt(errInputId, "Unknown operation.");
        }

        mxFree((void *)dimB);
        mxGPUDestroyGPUArray(B);
    }

    mxFree((void *)dimA);
    mxGPUDestroyGPUArray(A);
}
